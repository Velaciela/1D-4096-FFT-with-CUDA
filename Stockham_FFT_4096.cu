#include "hip/hip_runtime.h"
﻿


#include <stdio.h>
#include <string.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <cudaProfiler.h>

#include <hip/device_functions.h>
#include <>

#include <hip/hip_texture_types.h>
#include <>



#define checkCuda( fn ) do { \
		hipError_t error = (fn); \
		if ( hipSuccess != error ) { \
			const char* errstr = hipGetErrorString(error); \
			printf("%s returned error %s (code %d), line(%d)\n", #fn , errstr, error, __LINE__);\
			exit(EXIT_FAILURE); \
																		} \
																				} while (0)

typedef float2 Complex;
#define my_Pi 3.1415926535898 


int SARNRN = 8192;
int SARNAN = 4096;

size_t ADSampleComplx = SARNRN * SARNAN * sizeof(Complex);


#define IFFT_ON  1
#define IFFT_OFF 0

#define IFFT_DIV_N_ON  1
#define IFFT_DIV_N_OFF 0

#define FFTSHIFT_FRONT_ON  1
#define FFTSHIFT_FRONT_OFF 0

#define FFTSHIFT_BACK_ON  1
#define FFTSHIFT_BACK_OFF 0


extern "C"
__global__ void __launch_bounds__(256) Stockham_FFT_4096_SM(Complex *data_in, int ifft_flag, int ifft_divideN_flag, int fftshift_front, int fftshift_back);



int main()
{
	
	hipProfilerStart();

	printf("begin time is %ld \r\n", clock());

	//FILE* fp;
	float ms;

	//--------------------------------------------------------------
	//--------------------------SYS INIT----------------------------
	//--------------------------------------------------------------
#pragma region SYS INIT



	//---CPU MALLOC--------
#pragma region CPU MALLOC  	


	Complex *HostPinnedMemory;
	checkCuda(hipHostAlloc((void **)&HostPinnedMemory, ADSampleComplx, hipHostMallocDefault));

	printf("CPU MALLOC time is %ld \r\n", clock());

#pragma endregion
	//---CPU MALLOC--------

	//---GPU MALLOC--------
#pragma region GPU MALLOC  	

	//unsigned int FreeRamSpacePointerCounter = 0;

	Complex *d_ADcomplx;
	Complex *d_tmp;
	int *d_bitrevoder_128_table;
	int *d_bitrevoder_256_table;

	checkCuda(hipMalloc((void **)&d_ADcomplx, ADSampleComplx));        //256MB
	checkCuda(hipMalloc((void **)&d_tmp, ADSampleComplx));             //256MB

	

	printf("GPU MALLOC time is %ld \r\n", clock());
#pragma endregion
	//---GPU MALLOC--------

	//note:　FFT 配置 初始化 会占用 显存 同等大小的空间
	//---FFT PLAN----------
#pragma region FFT PLAN  


	int RANK = 1;   //1-D FFT
	int NX = 4096;//8192
	int BATCH = 8192 ;//4096
	int iembed = 4096;
	int istride = 1;   //连续无间隔
	int idist = 4096;  //the distance between the first element of two consecutive signals in a batch of the input data
	int oembed = 4096;
	int ostride = 1;
	int odist = 4096;


	hipfftHandle planRow;

	//hipfftPlanMany(&plan, RANK, NX, &iembed, istride, idist,&oembed, ostride, odist, HIPFFT_C2C, BATCH);

	//plan creat : 130ms
	if (hipfftPlanMany(&planRow, RANK, &NX, &iembed, istride, idist, &oembed, ostride, odist, HIPFFT_C2C, BATCH) != HIPFFT_SUCCESS){
		fprintf(stderr, "CUFFT error: Plan creation failed");
		return 0;
	}

#pragma endregion
	//---FFT PLAN----------


	//---PIC INIT----------

#pragma endregion
	//--------------------------------------------------------------
	//--------------------------SYS INIT----------------------------
	//--------------------------------------------------------------


	//------------------------ RUN CYCLE ---------------------------
	//--------------------------------------------------------------
	//--------------------------------------------------------------

#pragma region RUN CYCLE

	//cv::namedWindow("Display", CV_WINDOW_AUTOSIZE);



		//--------------------------------------------------------------
		//-------------------------DATA INPUT---------------------------
		//--------------------------------------------------------------

#pragma region DATA INPUT

		//---ADSample----------
#pragma region ADSample 


	for (int i = 0; i < SARNRN; i++)
	{
		for (int j = 0; j < SARNAN; j++)
		{
			HostPinnedMemory[i * SARNAN + j].x = j;//%256;
			HostPinnedMemory[i * SARNAN + j].y = j;//%256;
		}
	}
	

	checkCuda(hipMemcpy((Complex *)d_ADcomplx, (Complex *)HostPinnedMemory, ADSampleComplx, hipMemcpyHostToDevice));//36ms拷贝

	if (hipDeviceSynchronize() != hipSuccess){ fprintf(stderr, "Cuda error: Failed to synchronize\n"); return 0; }
	printf("AD Sample Input done %ld \r\n", clock());

	//--------------------AD Sample Transpose--------------------


#pragma endregion
		//---ADSample----------


		//GPU TIMER
		hipEvent_t startEvent, stopEvent;
		checkCuda(hipEventCreate(&startEvent));
		checkCuda(hipEventCreate(&stopEvent));
		
		checkCuda(hipEventRecord(startEvent, 0));


		Stockham_FFT_4096_SM<< <8192, 256 >> >(d_ADcomplx, IFFT_OFF, IFFT_DIV_N_OFF, FFTSHIFT_FRONT_OFF, FFTSHIFT_BACK_OFF);
	
		
		/*if (hipfftExecC2C(planRow, (hipfftComplex *)d_ADcomplx, (hipfftComplex *)d_ADcomplx, HIPFFT_FORWARD) != HIPFFT_SUCCESS ){
			fprintf(stderr, "CUFFT error: ExecC2C Forward failed");
			return;
		}
		if (hipDeviceSynchronize() != hipSuccess){ fprintf(stderr, "Cuda error: Failed to synchronize\n"); return; }*/



		checkCuda(hipEventRecord(stopEvent, 0));
		checkCuda(hipEventSynchronize(stopEvent));
		checkCuda(hipEventElapsedTime(&ms, startEvent, stopEvent));
		printf("!!!!!!!!!!1!!!!!!!!!! %f \r\n", ms);


		checkCuda(hipMemcpy((Complex *)HostPinnedMemory, (Complex *)( d_ADcomplx ), ADSampleComplx, hipMemcpyDeviceToHost));//36ms拷贝


		


		char txtdataFileName[1024];
		FILE * TxtWriter;


		sprintf(txtdataFileName, "stockham_FFT4096.txt");//F:/RDSAR/RDSAR/txtdata/z_matrix/
		TxtWriter = fopen(txtdataFileName, "wb");
		if (TxtWriter == 0)
		{
		printf("txt write creat error \r\n");
		return 0;
		}

		for (int i = 0; i < SARNAN; i++)
		{
			fprintf(TxtWriter, "%f \r\n", HostPinnedMemory[i].x);
			fprintf(TxtWriter, "%f \r\n", HostPinnedMemory[i].y);
		
		}

		fclose(TxtWriter);
		
		


#pragma endregion



	hipProfilerStop();


#pragma endregion


	//---CPU FREE-----------
#pragma region CPU FREE

	checkCuda(hipHostFree(HostPinnedMemory));

#pragma endregion
	//---CPU FREE-----------	

	//---GPU FREE-----------
#pragma region GPU FREE	

	hipfftDestroy(planRow);
	checkCuda(hipFree(d_ADcomplx));
	checkCuda(hipFree(d_tmp));

#pragma endregion
	//---GPU FREE-----------		

	//exit(EXIT_SUCCESS);
	return 0;
}







static __device__ inline Complex exp_calcu(float data_in)
{
    Complex data_out;

    data_out.x = __cosf(data_in);
    data_out.y = __sinf(data_in);
	//__sincosf(data_in, (&data_out.x),(&data_out.y));

    return data_out;
}

static __device__  inline Complex ComplexMul(Complex a, Complex b)
{
    Complex c;

    c.x = a.x*b.x - a.y*b.y;
    c.y = a.x*b.y + a.y*b.x;

    return c;
}

static __device__  inline Complex ComplexAdd(Complex a, Complex b)
{
    Complex c;

    c.x = a.x + b.x;
    c.y = a.y + b.y;

    return c;
}

static __device__  inline Complex ComplexSub(Complex a, Complex b)
{
    Complex c;

    c.x = a.x - b.x;
    c.y = a.y - b.y;

    return c;
}


static __device__ inline int StockhamR4_block(Complex *a, Complex *b, Complex *c, Complex *d, Complex *wi, float wi_c, int index)
{

    Complex tmp;
    Complex alpha, beta, gamma, delta;

    alpha = *a;                     *wi = exp_calcu(index * wi_c);  tmp = *wi;
    beta  = ComplexMul(*wi, *b);    *wi = ComplexMul(*wi, *wi);
    gamma = ComplexMul(*wi, *c);    *wi = ComplexMul(tmp, *wi);
    delta = ComplexMul(*wi, *d);

    tmp   = alpha;
    alpha = ComplexAdd(alpha, gamma);
    gamma = ComplexSub(tmp,   gamma);
    tmp   = beta;
    beta  = ComplexAdd(beta, delta);
    delta = ComplexSub(tmp,  delta);
    //tau3*wi(0,1)
    tmp.x   = delta.x;
    delta.x = -delta.y;
    delta.y = tmp.x;

    *a = ComplexAdd(alpha,  beta);
    *b = ComplexSub(gamma, delta);
    *c = ComplexSub(alpha,  beta);
    *d = ComplexAdd(gamma, delta);

    return 0;

}


static __device__ inline void Stockham_4096_block
(
    Complex *r1,  Complex *r2,  Complex *r3,  Complex *r4,
    Complex *r5,  Complex *r6,  Complex *r7,  Complex *r8,
    Complex *r9,  Complex *r10, Complex *r11, Complex *r12,
    Complex *r13, Complex *r14, Complex *r15, Complex *r16,
    Complex *data_shared
)
{

    //---------------------init-------------------------
    int bx  = blockIdx.x;
    int tid = threadIdx.x;

    int index = 0;

    float wi_c;
    Complex wi;

    const float FFT_Pi = 3.14159265359;

    ////Level in 4////////////////////////////////////////////////////////////////

    wi_c = -2*FFT_Pi/4;
    StockhamR4_block(r1,  r2,  r3,  r4,  &wi, wi_c, 0);
    StockhamR4_block(r5,  r6,  r7,  r8,  &wi, wi_c, 0);
    StockhamR4_block(r9,  r10, r11, r12, &wi, wi_c, 0);
    StockhamR4_block(r13, r14, r15, r16, &wi, wi_c, 0);

    data_shared[tid + 1024*0 + 256*0] = *r1;
    data_shared[tid + 1024*1 + 256*0] = *r2;
    data_shared[tid + 1024*2 + 256*0] = *r3;
    data_shared[tid + 1024*3 + 256*0] = *r4;
    data_shared[tid + 1024*0 + 256*1] = *r5;
    data_shared[tid + 1024*1 + 256*1] = *r6;
    data_shared[tid + 1024*2 + 256*1] = *r7;
    data_shared[tid + 1024*3 + 256*1] = *r8;
    data_shared[tid + 1024*0 + 256*2] = *r9;
    data_shared[tid + 1024*1 + 256*2] = *r10;
    data_shared[tid + 1024*2 + 256*2] = *r11;
    data_shared[tid + 1024*3 + 256*2] = *r12;
    data_shared[tid + 1024*0 + 256*3] = *r13;
    data_shared[tid + 1024*1 + 256*3] = *r14;
    data_shared[tid + 1024*2 + 256*3] = *r15;
    data_shared[tid + 1024*3 + 256*3] = *r16;
    __syncthreads();

    ////Level 16 64 256 1024///////////////////////////////////////////////////

    for(int i=1;i<=64;i=i*4)
    {
        int inexd_tmp = 256/i;

        index = (tid&(256 - inexd_tmp))*4 + tid%inexd_tmp;

        *r1  = data_shared[index + 1024*0 + inexd_tmp*0];
        *r2  = data_shared[index + 1024*0 + inexd_tmp*1];
        *r3  = data_shared[index + 1024*0 + inexd_tmp*2];
        *r4  = data_shared[index + 1024*0 + inexd_tmp*3];
        *r5  = data_shared[index + 1024*1 + inexd_tmp*0];
        *r6  = data_shared[index + 1024*1 + inexd_tmp*1];
        *r7  = data_shared[index + 1024*1 + inexd_tmp*2];
        *r8  = data_shared[index + 1024*1 + inexd_tmp*3];
        *r9  = data_shared[index + 1024*2 + inexd_tmp*0];
        *r10 = data_shared[index + 1024*2 + inexd_tmp*1];
        *r11 = data_shared[index + 1024*2 + inexd_tmp*2];
        *r12 = data_shared[index + 1024*2 + inexd_tmp*3];
        *r13 = data_shared[index + 1024*3 + inexd_tmp*0];
        *r14 = data_shared[index + 1024*3 + inexd_tmp*1];
        *r15 = data_shared[index + 1024*3 + inexd_tmp*2];
        *r16 = data_shared[index + 1024*3 + inexd_tmp*3];


        wi_c = -2*FFT_Pi/(16*i);
        index = tid / inexd_tmp;
        StockhamR4_block(r1,  r2,  r3,  r4,  &wi, wi_c, index + i*0);
        StockhamR4_block(r5,  r6,  r7,  r8,  &wi, wi_c, index + i*1);
        StockhamR4_block(r9,  r10, r11, r12, &wi, wi_c, index + i*2);
        StockhamR4_block(r13, r14, r15, r16, &wi, wi_c, index + i*3);

		__syncthreads();

        data_shared[tid + 1024*0 + 256*0] = *r1;
        data_shared[tid + 1024*1 + 256*0] = *r2;
        data_shared[tid + 1024*2 + 256*0] = *r3;
        data_shared[tid + 1024*3 + 256*0] = *r4;
        data_shared[tid + 1024*0 + 256*1] = *r5;
        data_shared[tid + 1024*1 + 256*1] = *r6;
        data_shared[tid + 1024*2 + 256*1] = *r7;
        data_shared[tid + 1024*3 + 256*1] = *r8;
        data_shared[tid + 1024*0 + 256*2] = *r9;
        data_shared[tid + 1024*1 + 256*2] = *r10;
        data_shared[tid + 1024*2 + 256*2] = *r11;
        data_shared[tid + 1024*3 + 256*2] = *r12;
        data_shared[tid + 1024*0 + 256*3] = *r13;
        data_shared[tid + 1024*1 + 256*3] = *r14;
        data_shared[tid + 1024*2 + 256*3] = *r15;
        data_shared[tid + 1024*3 + 256*3] = *r16;
        __syncthreads();

    }

    ////Level 4096//////////////////////////////////////////////////////////

    index=tid*4;

    *r1  = data_shared[index + 1024*0 + 0];
    *r2  = data_shared[index + 1024*0 + 1];
    *r3  = data_shared[index + 1024*0 + 2];
    *r4  = data_shared[index + 1024*0 + 3];
    *r5  = data_shared[index + 1024*1 + 0];
    *r6  = data_shared[index + 1024*1 + 1];
    *r7  = data_shared[index + 1024*1 + 2];
    *r8  = data_shared[index + 1024*1 + 3];
    *r9  = data_shared[index + 1024*2 + 0];
    *r10 = data_shared[index + 1024*2 + 1];
    *r11 = data_shared[index + 1024*2 + 2];
    *r12 = data_shared[index + 1024*2 + 3];
    *r13 = data_shared[index + 1024*3 + 0];
    *r14 = data_shared[index + 1024*3 + 1];
    *r15 = data_shared[index + 1024*3 + 2];
    *r16 = data_shared[index + 1024*3 + 3];

    wi_c  = -2*FFT_Pi/4096;
    index = tid;
    StockhamR4_block(r1,  r2,  r3,  r4,  &wi, wi_c, index + 256*0);
    StockhamR4_block(r5,  r6,  r7,  r8,  &wi, wi_c, index + 256*1);
    StockhamR4_block(r9,  r10, r11, r12, &wi, wi_c, index + 256*2);
    StockhamR4_block(r13, r14, r15, r16, &wi, wi_c, index + 256*3);
	__syncthreads();

    ////end//////////////////////////////////////////////////////////////////
}



//////////////////////////////////////////////////////////////////
////                                                          ////
////  Stockham_FFT    N=4096                                  ////
////                                                          ////
////  all in one kernel , no need of temporary space          ////
////                                                          ////
////                                     Date:                ////
////                                            2016-10-16    ////
//////////////////////////////////////////////////////////////////


extern "C"
__global__ void __launch_bounds__(256) Stockham_FFT_4096_SM(Complex *data_in, int ifft_flag, int ifft_divideN_flag, int fftshift_front, int fftshift_back)
{
    Complex rA1, rA2, rA3, rA4, rA5, rA6, rA7, rA8, rA9, rA10, rA11, rA12, rA13, rA14, rA15, rA16;

    __shared__ Complex data_in_shared_A[4096];

    int bx  = blockIdx.x;
    int tid = threadIdx.x;

    float wi_c;
    Complex wi;

    int index = 0;
    int index_in = bx * 4096 + tid;

    if(fftshift_front==1)
    {
        rA1  = data_in[index_in + 1024*2 + 256*0];
        rA2  = data_in[index_in + 1024*3 + 256*0];
        rA3  = data_in[index_in + 1024*0 + 256*0];
        rA4  = data_in[index_in + 1024*1 + 256*0];
        rA5  = data_in[index_in + 1024*2 + 256*1];
        rA6  = data_in[index_in + 1024*3 + 256*1];
        rA7  = data_in[index_in + 1024*0 + 256*1];
        rA8  = data_in[index_in + 1024*1 + 256*1];
        rA9  = data_in[index_in + 1024*2 + 256*2];
        rA10 = data_in[index_in + 1024*3 + 256*2];
        rA11 = data_in[index_in + 1024*0 + 256*2];
        rA12 = data_in[index_in + 1024*1 + 256*2];
        rA13 = data_in[index_in + 1024*2 + 256*3];
        rA14 = data_in[index_in + 1024*3 + 256*3];
        rA15 = data_in[index_in + 1024*0 + 256*3];
        rA16 = data_in[index_in + 1024*1 + 256*3];
    }
    else
    {
        rA1  = data_in[index_in + 1024*0 + 256*0];
        rA2  = data_in[index_in + 1024*1 + 256*0];
        rA3  = data_in[index_in + 1024*2 + 256*0];
        rA4  = data_in[index_in + 1024*3 + 256*0];
        rA5  = data_in[index_in + 1024*0 + 256*1];
        rA6  = data_in[index_in + 1024*1 + 256*1];
        rA7  = data_in[index_in + 1024*2 + 256*1];
        rA8  = data_in[index_in + 1024*3 + 256*1];
        rA9  = data_in[index_in + 1024*0 + 256*2];
        rA10 = data_in[index_in + 1024*1 + 256*2];
        rA11 = data_in[index_in + 1024*2 + 256*2];
        rA12 = data_in[index_in + 1024*3 + 256*2];
        rA13 = data_in[index_in + 1024*0 + 256*3];
        rA14 = data_in[index_in + 1024*1 + 256*3];
        rA15 = data_in[index_in + 1024*2 + 256*3];
        rA16 = data_in[index_in + 1024*3 + 256*3];
    }
    //Move your code of previous kernel here , if possible.-----------------------------





    //---------------------------------------------------------------------------------

    if(ifft_flag == 1)
    {
        rA1.y  = -rA1.y ;
        rA2.y  = -rA2.y ;
        rA3.y  = -rA3.y ;
        rA4.y  = -rA4.y ;
        rA5.y  = -rA5.y ;
        rA6.y  = -rA6.y ;
        rA7.y  = -rA7.y ;
        rA8.y  = -rA8.y ;
        rA9.y  = -rA9.y ;
        rA10.y = -rA10.y;
        rA11.y = -rA11.y;
        rA12.y = -rA12.y;
        rA13.y = -rA13.y;
        rA14.y = -rA14.y;
        rA15.y = -rA15.y;
        rA16.y = -rA16.y;
    }


    Stockham_4096_block(&rA1, &rA2, &rA3, &rA4, &rA5, &rA6, &rA7, &rA8,
                        &rA9, &rA10, &rA11, &rA12, &rA13, &rA14, &rA15, &rA16,
                        data_in_shared_A);

    if(ifft_flag == 1)
    {
        rA1.y  = -rA1.y ;
        rA2.y  = -rA2.y ;
        rA3.y  = -rA3.y ;
        rA4.y  = -rA4.y ;
        rA5.y  = -rA5.y ;
        rA6.y  = -rA6.y ;
        rA7.y  = -rA7.y ;
        rA8.y  = -rA8.y ;
        rA9.y  = -rA9.y ;
        rA10.y = -rA10.y;
        rA11.y = -rA11.y;
        rA12.y = -rA12.y;
        rA13.y = -rA13.y;
        rA14.y = -rA14.y;
        rA15.y = -rA15.y;
        rA16.y = -rA16.y;
    }

    if( ifft_divideN_flag && ifft_flag )
    {
        rA1.x  = rA1.x  / 4096 ;
        rA2.x  = rA2.x  / 4096 ;
        rA3.x  = rA3.x  / 4096 ;
        rA4.x  = rA4.x  / 4096 ;
        rA5.x  = rA5.x  / 4096 ;
        rA6.x  = rA6.x  / 4096 ;
        rA7.x  = rA7.x  / 4096 ;
        rA8.x  = rA8.x  / 4096 ;
        rA9.x  = rA9.x  / 4096 ;
        rA10.x = rA10.x / 4096 ;
        rA11.x = rA11.x / 4096 ;
        rA12.x = rA12.x / 4096 ;
        rA13.x = rA13.x / 4096 ;
        rA14.x = rA14.x / 4096 ;
        rA15.x = rA15.x / 4096 ;
        rA16.x = rA16.x / 4096 ;

        rA1.y  = rA1.y  / 4096 ;
        rA2.y  = rA2.y  / 4096 ;
        rA3.y  = rA3.y  / 4096 ;
        rA4.y  = rA4.y  / 4096 ;
        rA5.y  = rA5.y  / 4096 ;
        rA6.y  = rA6.y  / 4096 ;
        rA7.y  = rA7.y  / 4096 ;
        rA8.y  = rA8.y  / 4096 ;
        rA9.y  = rA9.y  / 4096 ;
        rA10.y = rA10.y / 4096 ;
        rA11.y = rA11.y / 4096 ;
        rA12.y = rA12.y / 4096 ;
        rA13.y = rA13.y / 4096 ;
        rA14.y = rA14.y / 4096 ;
        rA15.y = rA15.y / 4096 ;
        rA16.y = rA16.y / 4096 ;
    }


    //Move your code of next kernel here , if possible.-----------------------------





    //-----------------------------------------------------------------------------

    if(fftshift_back==1)
    {
        data_in[index_in + 1024*2 + 256*0] = rA1 ;
        data_in[index_in + 1024*3 + 256*0] = rA2 ;
        data_in[index_in + 1024*0 + 256*0] = rA3 ;
        data_in[index_in + 1024*1 + 256*0] = rA4 ;
        data_in[index_in + 1024*2 + 256*1] = rA5 ;
        data_in[index_in + 1024*3 + 256*1] = rA6 ;
        data_in[index_in + 1024*0 + 256*1] = rA7 ;
        data_in[index_in + 1024*1 + 256*1] = rA8 ;
        data_in[index_in + 1024*2 + 256*2] = rA9 ;
        data_in[index_in + 1024*3 + 256*2] = rA10;
        data_in[index_in + 1024*0 + 256*2] = rA11;
        data_in[index_in + 1024*1 + 256*2] = rA12;
        data_in[index_in + 1024*2 + 256*3] = rA13;
        data_in[index_in + 1024*3 + 256*3] = rA14;
        data_in[index_in + 1024*0 + 256*3] = rA15;
        data_in[index_in + 1024*1 + 256*3] = rA16;
    }
    else
    {
        data_in[index_in + 1024*0 + 256*0] = rA1 ;
        data_in[index_in + 1024*1 + 256*0] = rA2 ;
        data_in[index_in + 1024*2 + 256*0] = rA3 ;
        data_in[index_in + 1024*3 + 256*0] = rA4 ;
        data_in[index_in + 1024*0 + 256*1] = rA5 ;
        data_in[index_in + 1024*1 + 256*1] = rA6 ;
        data_in[index_in + 1024*2 + 256*1] = rA7 ;
        data_in[index_in + 1024*3 + 256*1] = rA8 ;
        data_in[index_in + 1024*0 + 256*2] = rA9 ;
        data_in[index_in + 1024*1 + 256*2] = rA10;
        data_in[index_in + 1024*2 + 256*2] = rA11;
        data_in[index_in + 1024*3 + 256*2] = rA12;
        data_in[index_in + 1024*0 + 256*3] = rA13;
        data_in[index_in + 1024*1 + 256*3] = rA14;
        data_in[index_in + 1024*2 + 256*3] = rA15;
        data_in[index_in + 1024*3 + 256*3] = rA16;
    }
}